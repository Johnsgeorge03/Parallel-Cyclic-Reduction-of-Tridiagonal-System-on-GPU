#include "hip/hip_runtime.h"
#include<iostream>
#include<iomanip>
#include<math.h> 
#include<chrono>
#include<float.h>

/*
 * Parallel Cyclic Reduction on GPU 
 * for solving Ax = b where A is tridiagonal and 
 * only the non-zero elements of A are stored. 
 * Compile using 
 * nvcc -g -G -c "%f"
 * build using
 * nvcc -g -G  -o "%e" "%f"
 *
 */

using namespace std;
using namespace std::chrono;
const int N = pow(2, 12);
const int threads_per_block = 256;

__global__ void kernel(double *d_F, double *d_D, double *d_Dsub, double *d_Dsup, \\
					   double *d_Ft, double *d_Dt, double *d_Dsubt, double *d_Dsupt,
					   double *d_y, int N, int i, int stride)
{
	int id 	   = blockIdx.x*blockDim.x + threadIdx.x;
	int index1 = id - stride;
	int index2 = id + stride;
	double gamma = 0.0, alpha = 0.0;
	
	if(id < N){
		if (index1 < 0){
			gamma   		= -d_Dsup[id]/d_D[index2];
			d_Dt[id]		= d_D[id] + d_Dsub[index2]*gamma;
			d_Dsupt[id]    	= d_Dsup[index2]*gamma;
			d_Ft[id]    	= d_F[id] + gamma*d_F[index2];
		}
			
		else if (index2 > N - 1 ){
			alpha   		= -d_Dsub[id]/d_D[index1];
			d_Dsubt[id]	    = d_Dsub[index1]*(alpha);
			d_Dt[id] 	    = d_D[id] + d_Dsup[index1]*alpha;
			d_Ft[id]   	    = d_F[id] + alpha*d_F[index1];
		}
			
		else{
			alpha   = -d_Dsub[id]/d_D[index1];
			gamma   = -d_Dsup[id]/d_D[index2];
			
			d_Dsubt[id]	    = d_Dsub[index1]*(alpha);
			d_Dt[id]    	= d_D[id] + (d_Dsup[index1]*alpha + d_Dsub[index2]*gamma);
			d_Dsupt[id]     = d_Dsup[index2]*gamma;
			d_Ft[id]        = d_F[id] + (alpha*d_F[index1] + gamma*d_F[index2]);
		}
		
		d_y[id] = d_Ft[id]/d_Dt[id];
		
	}
	
}


int main()
{
	/* Memory Allocation */
	double *y 		= (double*) malloc(N*sizeof(double));
	double *ytrue 	= (double*) malloc(N*sizeof(double));
	double dx 		= 1.0/(N + 1);

	for(int i = 0;i<N;i++){
		y[i] 		= 0.0;
		ytrue[i] 	= double(-100.0*(i+1)*dx + 373.15);
	}	

	double *F    = (double*) malloc( N* sizeof(double));
	double *D    = (double*) malloc( N* sizeof(double));
	double *Dsub = (double*) malloc( N* sizeof(double));
	double *Dsup = (double*) malloc( N* sizeof(double));
	
	
	double *d_F, *d_D, *d_Dsub, *d_Dsup, \\
			*d_Ft, *d_Dt, *d_Dsubt, *d_Dsupt, *d_y;
	
	for(int i=0;i<N;i++){
		D[i]    = -2.0;
		Dsub[i] = 1.0;
		Dsup[i] = 1.0;
		F[i]    = 0.0;
	}
	F[0]      = -373.15;
	F[N - 1]  = -273.15;
	

	Dsub[0]   = 0.0;
	Dsup[N-1] = 0.0;
	
	
	int num_blocks 			= max(N/threads_per_block, 1);
	cout<<"num_blocks = "<<num_blocks<<"\n"<<endl;
	dim3 grid_size (num_blocks);
	
	auto start_mem_transfer = high_resolution_clock::now();
	
	hipMalloc(&d_F , N*sizeof(double));
	hipMalloc(&d_D , N*sizeof(double));
	hipMalloc(&d_Dsub , N*sizeof(double));
	hipMalloc(&d_Dsup, N*sizeof(double));
	
	hipMalloc(&d_Ft , N*sizeof(double));
	hipMalloc(&d_Dt , N*sizeof(double));
	hipMalloc(&d_Dsubt , N*sizeof(double));
	hipMalloc(&d_Dsupt, N*sizeof(double));
	hipMalloc(&d_y, N*sizeof(double));
	
	hipMemcpy(d_F, F, N*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_D, D, N*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_Dsub, Dsub, N*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_Dsup, Dsup, N*sizeof(double), hipMemcpyHostToDevice);
	
	auto stop_mem_transfer = high_resolution_clock::now();
	auto durationt1 = duration_cast<microseconds>(stop_mem_transfer - start_mem_transfer);
	
	
	auto start = high_resolution_clock::now();
	
	for(int i = 0; i < log2(N); i++){
		int stride = pow(2, i);
		kernel<<<grid_size , threads_per_block>>>\\
		(d_F, d_D, d_Dsub, d_Dsup, d_Ft, d_Dt, d_Dsubt, d_Dsupt, \\
											d_y,  N, i, stride);
											
		hipDeviceSynchronize();
		double* t1 = d_Ft;
		double* t2 = d_Dt;
		double* t3 = d_Dsubt;
		double* t4 = d_Dsupt;
		
		d_Ft 	= d_F;
		d_Dt 	= d_D;
		d_Dsubt = d_Dsub;
		d_Dsupt = d_Dsup;
		
		d_F 	= t1;
		d_D 	= t2;
		d_Dsub  = t3;
		d_Dsup  = t4;
	}
	//cout<<"first trans :"<< durationt1.count() <<endl;
	
	auto stop = high_resolution_clock::now();
	auto duration = duration_cast<microseconds>(stop - start);
	cout << "kernel compution :"<<duration.count() << endl;
	cout<< "---------------------------------------------" << endl;
	
	auto start_mem_transfer2 = high_resolution_clock::now();
	hipMemcpy(y, d_y, N*sizeof(double), hipMemcpyDeviceToHost);
	auto stop_mem_transfer2 = high_resolution_clock::now();
	auto durationt2 = duration_cast<microseconds>(stop_mem_transfer2 - start_mem_transfer2);
	auto total_mem_trans = durationt1 + durationt2;
	
	cout << "Mem transfer and allocation :"<<total_mem_trans.count() << endl;
	cout<< "---------------------------------------------" << endl;
	
	//hipMemcpy(Dsup, d_Dsup, N*sizeof(double), hipMemcpyDeviceToHost);
	double err = DBL_MIN; 
	double temp;
	for(int i=0;i<N;i++){
		temp = abs( y[i] - ytrue[i]);
		if (temp > err) err  = temp;
	}
	cout<<"Max_error  :"<< err << endl;
	
	//~ for(int i=0;i<N;i++){
		//~ cout <<"i : "<< i << "--  " << y[i] << " true :" << ytrue[i] << endl;
	//~ }	
	
	free(Dsub);
	free(D);
	free(Dsup);
	free(F);
	return 0;
}
